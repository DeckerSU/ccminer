#include "hip/hip_runtime.h"

#include <miner.h>
extern "C" {
#include <stdint.h>
#include <memory.h>
}
#define HARAKAS_RATE 32

#include <cuda_helper.h>

#define NPT 2
#define NBN 2


__global__ void verus_gpu_hash(uint32_t threads, uint32_t startNonce, uint32_t *resNonce);

__device__ void haraka512_full(unsigned char *out, const unsigned char *in);
__device__ void haraka512_perm(unsigned char *out, const unsigned char *in);
	

static uint32_t *d_nonces[MAX_GPUS];

__constant__ uint8_t blockhash_half[128];
__constant__ uint32_t ptarget[8];

__host__
void verus_init(int thr_id)
{
	
	CUDA_SAFE_CALL(hipMalloc(&d_nonces[thr_id], 2*sizeof(uint32_t)));
   
};


void verus_setBlock(void *blockf,const void *pTargetIn) 
{
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ptarget), pTargetIn, 8*sizeof(uint32_t), 0, hipMemcpyHostToDevice));
 	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(blockhash_half), blockf, 64*sizeof(uint8_t), 0, hipMemcpyHostToDevice));
};

__host__ 
void verus_hash(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t *resNonces)
{
	hipMemset(d_nonces[thr_id], 0xff, 2 * sizeof(uint32_t));
	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	verus_gpu_hash<<<grid, block>>>(threads, startNonce, d_nonces[thr_id]);
	hipDeviceSynchronize();
	hipMemcpy(resNonces, d_nonces[thr_id], 2*sizeof(uint32_t), hipMemcpyDeviceToHost);
 
};

__global__ 
void verus_gpu_hash(uint32_t threads, uint32_t startNonce, uint32_t *resNonce)
{
	uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;
	if (thread < threads)
	{
			uint32_t nounce = startNonce + thread;

			uint8_t hash_buf[64];
			uint8_t blockhash[64];
    
			memcpy(hash_buf,blockhash_half,128);
			memset(hash_buf + 32, 0x0,32);
			//memcpy(hash_buf + 32, (unsigned char *)&full_data + 1486 - 14, 15);
			((uint32_t *)&hash_buf)[8] = nounce;
  
			haraka512_full((unsigned char*)blockhash, (unsigned char*)hash_buf); // ( out, in)

			if (((uint64_t*)&blockhash)[3] < ((uint64_t*)&ptarget)[3]) { resNonce[0] = nounce;}   
    }
};

__device__ void memcpy_decker(unsigned char *dst, unsigned char *src, int len) {
    int i;
    for (i=0; i<len; i++) { dst[i] = src[i]; }
}

//__constant__ static const
__device__  unsigned char sbox[256] =
{ 0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe,
  0xd7, 0xab, 0x76, 0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4,
  0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0, 0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7,
  0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15, 0x04, 0xc7, 0x23, 0xc3,
  0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75, 0x09,
  0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3,
  0x2f, 0x84, 0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe,
  0x39, 0x4a, 0x4c, 0x58, 0xcf, 0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85,
  0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8, 0x51, 0xa3, 0x40, 0x8f, 0x92,
  0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2, 0xcd, 0x0c,
  0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19,
  0x73, 0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14,
  0xde, 0x5e, 0x0b, 0xdb, 0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2,
  0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79, 0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5,
  0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08, 0xba, 0x78, 0x25,
  0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
  0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86,
  0xc1, 0x1d, 0x9e, 0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e,
  0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf, 0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42,
  0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16 };

__device__  unsigned char smod[256] =
{ 0x00, 0x03, 0x06, 0x05, 0x0C, 0x0F, 0x0A, 0x09, 0x18, 0x1B, 0x1E, 0x1D, 0x14,
0x17, 0x12, 0x11, 0x30, 0x33, 0x36, 0x35, 0x3C, 0x3F, 0x3A, 0x39, 0x28, 0x2B,
0x2E, 0x2D, 0x24, 0x27, 0x22, 0x21, 0x60, 0x63, 0x66, 0x65, 0x6C, 0x6F, 0x6A,
0x69, 0x78, 0x7B, 0x7E, 0x7D, 0x74, 0x77, 0x72, 0x71, 0x50, 0x53, 0x56, 0x55,
0x5C, 0x5F, 0x5A, 0x59, 0x48, 0x4B, 0x4E, 0x4D, 0x44, 0x47, 0x42, 0x41, 0xC0,
0xC3, 0xC6, 0xC5, 0xCC, 0xCF, 0xCA, 0xC9, 0xD8, 0xDB, 0xDE, 0xDD, 0xD4, 0xD7,
0xD2, 0xD1, 0xF0, 0xF3, 0xF6, 0xF5, 0xFC, 0xFF, 0xFA, 0xF9, 0xE8, 0xEB, 0xEE,
0xED, 0xE4, 0xE7, 0xE2, 0xE1, 0xA0, 0xA3, 0xA6, 0xA5, 0xAC, 0xAF, 0xAA, 0xA9,
0xB8, 0xBB, 0xBE, 0xBD, 0xB4, 0xB7, 0xB2, 0xB1, 0x90, 0x93, 0x96, 0x95, 0x9C,
0x9F, 0x9A, 0x99, 0x88, 0x8B, 0x8E, 0x8D, 0x84, 0x87, 0x82, 0x81, 0x9B, 0x98,
0x9D, 0x9E, 0x97, 0x94, 0x91, 0x92, 0x83, 0x80, 0x85, 0x86, 0x8F, 0x8C, 0x89,
0x8A, 0xAB, 0xA8, 0xAD, 0xAE, 0xA7, 0xA4, 0xA1, 0xA2, 0xB3, 0xB0, 0xB5, 0xB6,
0xBF, 0xBC, 0xB9, 0xBA, 0xFB, 0xF8, 0xFD, 0xFE, 0xF7, 0xF4, 0xF1, 0xF2, 0xE3,
0xE0, 0xE5, 0xE6, 0xEF, 0xEC, 0xE9, 0xEA, 0xCB, 0xC8, 0xCD, 0xCE, 0xC7, 0xC4,
0xC1, 0xC2, 0xD3, 0xD0, 0xD5, 0xD6, 0xDF, 0xDC, 0xD9, 0xDA, 0x5B, 0x58, 0x5D,
0x5E, 0x57, 0x54, 0x51, 0x52, 0x43, 0x40, 0x45, 0x46, 0x4F, 0x4C, 0x49, 0x4A,
0x6B, 0x68, 0x6D, 0x6E, 0x67, 0x64, 0x61, 0x62, 0x73, 0x70, 0x75, 0x76, 0x7F,
0x7C, 0x79, 0x7A, 0x3B, 0x38, 0x3D, 0x3E, 0x37, 0x34, 0x31, 0x32, 0x23, 0x20,
0x25, 0x26, 0x2F, 0x2C, 0x29, 0x2A, 0x0B, 0x08, 0x0D, 0x0E, 0x07, 0x04, 0x01,
0x02, 0x13, 0x10, 0x15, 0x16, 0x1F, 0x1C, 0x19, 0x1A };


#define XT(x) (((x) << 1) ^ ((((x) >> 7) & 1) * 0x1b))

// Simulate _mm_aesenc_si128 instructions from AESNI
__device__  void aesenc(unsigned char *s,const unsigned char sharedMemory1[256])
{
    unsigned char i, t, u, v[4][4];
    for (i = 0; i < 16; ++i) {
        v[((i / 4) + 4 - (i%4) ) % 4][i % 4] = sharedMemory1[s[i]];
    }
    for (i = 0; i < 4; ++i) {
        t = v[i][0];
        u = v[i][0] ^ v[i][1] ^ v[i][2] ^ v[i][3];
        v[i][0] ^= u ^ XT(v[i][0] ^ v[i][1]);
        v[i][1] ^= u ^ XT(v[i][1] ^ v[i][2]);
        v[i][2] ^= u ^ XT(v[i][2] ^ v[i][3]);
        v[i][3] ^= u ^ XT(v[i][3] ^ t);
    }
    for (i = 0; i < 16; ++i) {
        s[i] = v[i / 4][i % 4]; // VerusHash have 0 rc vector
    }
}

__device__ void aesenc_double(unsigned char *s, const unsigned char sharedMemory1[256], const unsigned char sharedMemory2[256])
{
	unsigned char v0, v1, v2, v3, v4, v5, v6, v7;
	unsigned char t0, t1, t2;

	v0 = sharedMemory1[s[0]];
	v1 = sharedMemory1[s[5]];
	v2 = sharedMemory1[s[10]];
	v3 = sharedMemory1[s[15]];
	v4 = sharedMemory1[s[3]];
	v5 = sharedMemory1[s[2]];
	v6 = sharedMemory1[s[1]];

	t0 = v0 ^ v1;
	t1 = v1 ^ v2;
	t2 = v2 ^ v3;

	s[0] = sharedMemory2[t0] ^ v0 ^ t2;
	s[1] = sharedMemory2[t1] ^ t0 ^ v3;
	s[2] = sharedMemory2[t2] ^ v2 ^ t0;
	s[3] = sharedMemory2[v3 ^ v0] ^ v3 ^ t1;

	v0 = sharedMemory1[s[4]];
	v1 = sharedMemory1[s[9]];
	v2 = sharedMemory1[s[14]];
	v3 = v4;
	v4 = sharedMemory1[s[7]];
	v7 = sharedMemory1[s[6]];

	t0 = v0 ^ v1;
	t1 = v1 ^ v2;
	t2 = v2 ^ v3;

	s[4] = sharedMemory2[t0] ^ v0 ^ t2;
	s[5] = sharedMemory2[t1] ^ t0 ^ v3;
	s[6] = sharedMemory2[t2] ^ v2 ^ t0;
	s[7] = sharedMemory2[v3 ^ v0] ^ v3 ^ t1;

	v0 = sharedMemory1[s[8]];
	v1 = sharedMemory1[s[13]];
	v2 = v5;
	v3 = v4;
	v5 = sharedMemory1[s[11]];

	t0 = v0 ^ v1;
	t1 = v1 ^ v2;
	t2 = v2 ^ v3;

	s[8] = sharedMemory2[t0] ^ v0 ^ t2;
	s[9] = sharedMemory2[t1] ^ t0 ^ v3;

	s[10] = sharedMemory2[t2] ^ v2 ^ t0;
	s[11] = sharedMemory2[v3 ^ v0] ^ v3 ^ t1;

	v0 = sharedMemory1[s[12]];
	v1 = v6;
	v2 = v7;
	v3 = v5;

	t0 = v0 ^ v1;
	t1 = v1 ^ v2;
	t2 = v2 ^ v3;
	s[12] = sharedMemory2[t0] ^ v0 ^ t2;
	s[13] = sharedMemory2[t1] ^ t0 ^ v3;
	s[14] = sharedMemory2[t2] ^ v2 ^ t0;
	s[15] = sharedMemory2[v3 ^ v0] ^ v3 ^ t1;

	v0 = sharedMemory1[s[0]];
	v1 = sharedMemory1[s[5]];
	v2 = sharedMemory1[s[10]];
	v3 = sharedMemory1[s[15]];
	v4 = sharedMemory1[s[3]];
	v5 = sharedMemory1[s[2]];
	v6 = sharedMemory1[s[1]];

	t0 = v0 ^ v1;
	t1 = v1 ^ v2;
	t2 = v2 ^ v3;
	s[0] = sharedMemory2[t0] ^ v0 ^ t2;
	s[1] = sharedMemory2[t1] ^ t0 ^ v3;
	s[2] = sharedMemory2[t2] ^ v2 ^ t0;
	s[3] = sharedMemory2[v3 ^ v0] ^ v3 ^ t1;

	v0 = sharedMemory1[s[4]];
	v1 = sharedMemory1[s[9]];
	v2 = sharedMemory1[s[14]];
	v3 = v4;
	v4 = sharedMemory1[s[7]];
	v7 = sharedMemory1[s[6]];

	t0 = v0 ^ v1;
	t1 = v1 ^ v2;
	t2 = v2 ^ v3;
	s[4] = sharedMemory2[t0] ^ v0 ^ t2;
	s[5] = sharedMemory2[t1] ^ t0 ^ v3;
	s[6] = sharedMemory2[t2] ^ v2 ^ t0;
	s[7] = sharedMemory2[v3 ^ v0] ^ v3 ^ t1;

	v0 = sharedMemory1[s[8]];
	v1 = sharedMemory1[s[13]];
	v2 = v5;
	v3 = v4;
	v5 = sharedMemory1[s[11]];

	t0 = v0 ^ v1;
	t1 = v1 ^ v2;
	t2 = v2 ^ v3;
	s[8] = sharedMemory2[t0] ^ v0 ^ t2;
	s[9] = sharedMemory2[t1] ^ t0 ^ v3;
	s[10] = sharedMemory2[t2] ^ v2 ^ t0;
	s[11] = sharedMemory2[v3 ^ v0] ^ v3 ^ t1;

	v0 = sharedMemory1[s[12]];
	v1 = v6;
	v2 = v7;
	v3 = v5;

	t0 = v0 ^ v1;
	t1 = v1 ^ v2;
	t2 = v2 ^ v3;
	s[12] = sharedMemory2[t0] ^ v0 ^ t2;
	s[13] = sharedMemory2[t1] ^ t0 ^ v3;
	s[14] = sharedMemory2[t2] ^ v2 ^ t0;
	s[15] = sharedMemory2[v3 ^ v0] ^ v3 ^ t1;
}

// Simulate _mm_unpacklo_epi32
__device__ __forceinline__ void unpacklo32(unsigned char *t, unsigned char *a, unsigned char *b)
{
    unsigned char tmp[16];
    memcpy_decker(tmp, a, 4);
    memcpy_decker(tmp + 4, b, 4);
    memcpy_decker(tmp + 8, a + 4, 4);
    memcpy_decker(tmp + 12, b + 4, 4);
    memcpy_decker(t, tmp, 16);
}

// Simulate _mm_unpackhi_epi32
__device__ __forceinline__ void unpackhi32(unsigned char *t, unsigned char *a, unsigned char *b)
{
    unsigned char tmp[16];
    memcpy_decker(tmp, a + 8, 4);
    memcpy_decker(tmp + 4, b + 8, 4);
    memcpy_decker(tmp + 8, a + 12, 4);
    memcpy_decker(tmp + 12, b + 12, 4);
    memcpy_decker(t, tmp, 16);
}



__device__ void haraka512_perm(unsigned char *out, const unsigned char *in) 
{

    int i, j;
	__shared__ unsigned char sharedMemory1[256];
	
	if (threadIdx.x < 256) 
		sharedMemory1[threadIdx.x] = sbox[threadIdx.x];

    unsigned char s[64], tmp[16];
    memcpy_decker(s, (unsigned char *)in, 64);
#pragma unroll
    for (i = 0; i < 5; ++i) {
        // aes round(s)
		
			for (j = 0; j < 2; ++j) {

				aesenc(s, sharedMemory1);
				aesenc(s + 16, sharedMemory1);
				aesenc(s + 32, sharedMemory1);
				aesenc(s + 48, sharedMemory1);
			}
		
		unpacklo32(tmp, s, s + 16);
		
		unpackhi32(s, s, s + 16);
        unpacklo32(s + 16, s + 32, s + 48);
        unpackhi32(s + 32, s + 32, s + 48);
        unpacklo32(s + 48, s, s + 32);
        unpackhi32(s, s, s + 32);
        unpackhi32(s + 32, s + 16, tmp);
        unpacklo32(s + 16, s + 16, tmp);
    }

    memcpy_decker(out, s, 64);
}

/*__device__ void haraka512_full(unsigned char *out, const unsigned char *in)
{
    int i;

    unsigned char buf[64];
    haraka512_perm(buf, in);
    for (i = 0; i < 64; i++) {
        buf[i] = buf[i] ^ in[i];
    }

    memcpy_decker(out,      buf + 8, 8);
    memcpy_decker(out + 8,  buf + 24, 8);
    memcpy_decker(out + 16, buf + 32, 8);
    memcpy_decker(out + 24, buf + 48, 8);
}*/

__device__ void haraka512_full(unsigned char *out, const unsigned char *in) {

	__shared__ unsigned char sharedMemory1[256];
	__shared__ unsigned char sharedMemory2[256];

	if (threadIdx.x < 256)
		sharedMemory1[threadIdx.x] = sbox[threadIdx.x];
	if (threadIdx.x < 256)
		sharedMemory2[threadIdx.x] = smod[threadIdx.x];

	unsigned char s[64];
	uint32_t *sd = (uint32_t*)(&s[0]);
	memcpy_decker(s, (unsigned char *)in, 64);
	
	#pragma unroll
	for (int i = 0; i < 4; ++i) {

		aesenc_double(s, sharedMemory1, sharedMemory2);
		aesenc_double(s + 16, sharedMemory1, sharedMemory2);
		aesenc_double(s + 32, sharedMemory1, sharedMemory2);
		aesenc_double(s + 48, sharedMemory1, sharedMemory2);

		// mixing
		uint32_t t;

		t = sd[0];
		sd[0] = sd[3];
		sd[3] = sd[15];
		sd[15] = sd[14];
		sd[14] = sd[6];
		sd[6] = sd[12];
		sd[12] = sd[2];
		sd[2] = sd[7];
		sd[7] = sd[4];
		sd[4] = sd[8];
		sd[8] = sd[9];
		sd[9] = sd[1];
		sd[1] = sd[11];
		sd[11] = sd[5];
		sd[5] = t;

		t = sd[13];
		sd[13] = sd[10];
		sd[10] = t;
	}

	aesenc_double(s, sharedMemory1, sharedMemory2);
	aesenc_double(s + 16, sharedMemory1, sharedMemory2);
	aesenc_double(s + 32, sharedMemory1, sharedMemory2);
	aesenc_double(s + 48, sharedMemory1, sharedMemory2);

	uint32_t *outd = ((uint32_t *)out);
	uint32_t *ind = ((uint32_t *)in);

	//__syncthreads();
	*outd++ = sd[7] ^ ind[2];
	*outd++ = sd[15] ^ ind[3];
	*outd++ = sd[12] ^ ind[6];
	*outd++ = sd[4] ^ ind[7];
	*outd++ = sd[9] ^ ind[8];
	*outd++ = sd[1] ^ ind[9];
	*outd++ = sd[2] ^ ind[12];
	*outd = sd[10] ^ ind[13];
	//__syncthreads();


	/*

	// original variant

	int i, j;
	unsigned char buf[64];

	unsigned char s[64], tmp[16];
	memcpy_decker(s, (unsigned char *)in, 64);
	#pragma unroll
	for (i = 0; i < 5; ++i) {
		// aes round(s)

		for (j = 0; j < 2; ++j) {

			aesenc(s, sharedMemory1);
			aesenc(s + 16, sharedMemory1);
			aesenc(s + 32, sharedMemory1);
			aesenc(s + 48, sharedMemory1);
		}

		unpacklo32(tmp, s, s + 16);

		unpackhi32(s, s, s + 16);
		unpacklo32(s + 16, s + 32, s + 48);
		unpackhi32(s + 32, s + 32, s + 48);
		unpacklo32(s + 48, s, s + 32);
		unpackhi32(s, s, s + 32);
		unpackhi32(s + 32, s + 16, tmp);
		unpacklo32(s + 16, s + 16, tmp);
	}

	memcpy_decker(buf, s, 64);
	

	for (i = 0; i < 64; i++) {
		buf[i] = buf[i] ^ in[i];
	}

	memcpy_decker(out, buf + 8, 8);
	memcpy_decker(out + 8, buf + 24, 8);
	memcpy_decker(out + 16, buf + 32, 8);
	memcpy_decker(out + 24, buf + 48, 8);
	*/

}
